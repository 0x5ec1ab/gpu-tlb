#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

#define CHUNK0_SIZE (64L * 1024L * 1024L * 1024L * 1024L + 0x55554000000L)
#define CHUNK1_SIZE (41L * 1024L * 1024L * 1024L * 1024L + 0x0ffc8000000L)
#define STRIDE_SIZE (1L * 1024L * 1024L)

#define BASE_ADDR   0x700000000000
#define DUMMY_ADDR  0x7F0000000000

#define PAGE_NUM    4000
//#define PAGE_NUM    2000
#define WAIT_TIME   10000000000L // about 5 seconds on RTX3080

__global__ void 
loop(volatile uint64_t *page, uint64_t x)
{
  uint64_t y = x;
  volatile uint64_t *ptr;
  uint64_t clk0;
  uint64_t clk1;
  
  while (y == x) {
    for (ptr = (uint64_t *)page[0]; ptr != page; ptr = (uint64_t *)ptr[0])
      ++ptr[2];
    
    clk0 = clock64();
    clk1 = 0;
    while (clk1 < WAIT_TIME)
      clk1 = clock64() - clk0;
    
    y = ptr[1];
  }
}

__global__ void
put(uint64_t *page, uint64_t x1, uint64_t x2)
{
  page[0] = x1;
  page[1] = x2;
}

int 
main(int argc, char *argv[])
{
  uint8_t *chunk0 = NULL;
  uint8_t *chunk1 = NULL;
  uint8_t *base = NULL;
  uint64_t *list[PAGE_NUM];
  uint64_t *dummy = NULL;
  
  hipDeviceReset();
  
  // hoard a large address space
  hipMallocManaged(&chunk0, CHUNK0_SIZE);
  hipMallocManaged(&chunk1, CHUNK1_SIZE);
  
  base = (uint8_t *)BASE_ADDR;
  for (int i = 0; i < PAGE_NUM; ++i)
    list[i] = (uint64_t *)(base + i * STRIDE_SIZE);
  dummy = (uint64_t *)DUMMY_ADDR;
  
  for (int i = 0; i < PAGE_NUM; ++i)
    put<<<1, 1>>>(list[i], (uint64_t)list[(i + 1) % PAGE_NUM], 0xdeadbeef);
  put<<<1, 1>>>(dummy, 0, 0);
  hipDeviceSynchronize();
  
  loop<<<1, 1>>>(list[0], 0xdeadbeef);
  hipDeviceSynchronize();
  
  hipFree(chunk0);
  hipFree(chunk1);
}


