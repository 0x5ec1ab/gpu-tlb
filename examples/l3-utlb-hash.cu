#include <hip/hip_runtime.h>
#include <algorithm>
#include <cstdint>
#include <cstring>
#include <iostream>
#include <vector>

#define CHUNK0_SIZE (64L * 1024L * 1024L * 1024L * 1024L + 0x55554000000L)
#define CHUNK1_SIZE (41L * 1024L * 1024L * 1024L * 1024L + 0x0ffc8000000L)
#define STRIDE_SIZE (1L * 1024L * 1024L)

#define BASE_ADDR   0x700000000000
#define DUMMY_ADDR  0x7F0000000000

#define FILL_NUM    3000
#define WAIT_TIME   10000000000L // about 5 seconds on RTX3080

std::vector<int> l1_idx_vec = {
  0, 129, 258, 387,
  1, 128, 259, 386, 
  2, 131, 256, 385, 
  3, 130, 257, 384, 
};

// these numbers form an L2-uTLB eviction set for target 32 
std::vector<int> l2_idx_vec = {
  131628, 131757, 131886, 132015, 
  132128, 132257, 132386, 132515,
};

// these numbers form an L3-uTLB eviction set for target 32 (derived using l3-utlb-set)
std::vector<int> l3_idx_vec = {
  32607, 33185, 65246, 66338, 
  97373, 98979, 130524, 132644,
};

/*******************************************************************************
 * derive the L3-uTLB set selection hash function
 ******************************************************************************/
__global__ void 
loop(volatile uint64_t *chain, volatile uint64_t *evict, volatile uint64_t *fill, uint64_t x)
{
  uint64_t y = x;
  volatile uint64_t *ptr;
  volatile uint64_t *evt;
  uint64_t clk0 = 0;
  uint64_t clk1 = 0;
  
  for (ptr = (uint64_t *)fill[0]; ptr != fill; ptr = (uint64_t *)ptr[0])
    ++ptr[2];
  
  while (y == x) {
    ptr = chain;
    do {
      ++ptr[2];
      ptr = (uint64_t *)ptr[0];
      for (evt = (uint64_t *)evict[0]; evt != evict; evt = (uint64_t *)evt[0])
        ++evt[2];
    } while (ptr != chain);
    
    clk0 = clock64();
    clk1 = 0;
    while (clk1 < WAIT_TIME)
      clk1 = clock64() - clk0;
    
    y = chain[1];
  }
}

__global__ void
put(uint64_t *page, uint64_t x1, uint64_t x2)
{
  page[0] = x1;
  page[1] = x2;
}

int 
main(int argc, char *argv[])
{
  hipDeviceReset();
  
  // hoard a large address space
  uint8_t *chunk0 = NULL;
  uint8_t *chunk1 = NULL;
  hipMallocManaged(&chunk0, CHUNK0_SIZE);
  hipMallocManaged(&chunk1, CHUNK1_SIZE);
  
  std::vector<uint64_t *> chain;
  int target = std::stoi(argv[1]);
  chain.push_back((uint64_t *)(BASE_ADDR + target * STRIDE_SIZE));
  for (auto i : l3_idx_vec)
    chain.push_back((uint64_t *)(BASE_ADDR + i * STRIDE_SIZE));
  std::vector<uint64_t *> evict;
  for (auto i : l1_idx_vec)
    evict.push_back((uint64_t *)(BASE_ADDR + i * STRIDE_SIZE));
  for (auto i : l2_idx_vec)
    evict.push_back((uint64_t *)(BASE_ADDR + i * STRIDE_SIZE));
  std::vector<uint64_t *> fill;
  for (int i = 1; i <= FILL_NUM; ++i)
    fill.push_back((uint64_t *)(DUMMY_ADDR + i * STRIDE_SIZE));
  uint64_t *dummy = (uint64_t *)DUMMY_ADDR;
  
  //++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
  // manipulate the last virtual address in chain
  int bit0 = argc >= 3 ? std::stoi(argv[2]) : -1;
  uint64_t mask0 = bit0 >= 0 ? 0x0000000000000001UL << bit0 : 0;
  int bit1 = argc == 4 ? std::stoi(argv[3]) : -1;
  uint64_t mask1 = bit1 >= 0 ? 0x0000000000000001UL << bit1 : 0;
  uint64_t addr = (uint64_t)chain.back();
  addr ^= mask0;
  addr ^= mask1;
  
  if (((uint8_t *)addr < chunk0 || (uint8_t *)addr >= chunk0 + CHUNK0_SIZE) && 
      ((uint8_t *)addr < chunk1 || (uint8_t *)addr >= chunk1 + CHUNK1_SIZE)) {
    std::cout << "out of scope: try another setting (e.g., BASE_ADDR or target)\n";
    return -1;
  }
  
  // an address in the set is generated
  for (size_t i = 0; i < chain.size() - 1; ++i) {
    if (addr == (uint64_t)chain[i])
      return 0;
  }
  
  for (auto temp : evict) {
    if (addr == (uint64_t)temp)
      while (1)
        ;
  }
  
  chain.back() = (uint64_t *)addr;
  //++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++  
  
  // do dummy first to ensure its physical address unchanged when changing inputs 
  put<<<1, 1>>>(dummy, 0, 0);
  for (size_t i = 0; i < chain.size(); ++i)
    put<<<1, 1>>>(chain[i], (uint64_t)chain[(i + 1) % chain.size()], 0xdeadbeef);
  for (size_t i = 0; i < evict.size(); ++i)
    put<<<1, 1>>>(evict[i], (uint64_t)evict[(i + 1) % evict.size()], 0);
  for (size_t i = 0; i < fill.size(); ++i)
    put<<<1, 1>>>(fill[i], (uint64_t)fill[(i + 1) % fill.size()], 0);
  hipDeviceSynchronize();
  
  loop<<<1, 1>>>(chain[0], evict[0], fill[0], 0xdeadbeef);
  hipDeviceSynchronize();
  
  hipFree(chunk0);
  hipFree(chunk1);
}


