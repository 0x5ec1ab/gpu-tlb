#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#define CHUNK0_SIZE (64L * 1024L * 1024L * 1024L * 1024L + 0x55554000000L)
#define CHUNK1_SIZE (41L * 1024L * 1024L * 1024L * 1024L + 0x0ffc8000000L)
#define STRIDE_SIZE (1L * 1024L * 1024L)

#define BASE_ADDR   0x700000000000
#define DUMMY_ADDR  0x7F0000000000

#define PAGE0_NUM   17
#define PAGE1_NUM   4000
#define WAIT_TIME   10000000000L // about 5 seconds on RTX3080

#define BLK_NUM     100
#define SHARED_MEM  (96 * 1024)
#define SMID0       0
#define SMID1       12 // IMPORTANT: SM0 and SM12 are in the same GPC on RTX3080

__global__ void 
loop(volatile uint64_t *page0, volatile uint64_t *page1, uint64_t x)
{
  uint64_t y = x;
  volatile uint64_t *ptr = NULL;
  uint64_t clk0 = 0;
  uint64_t clk1 = 0;
  uint32_t smid;
  
  asm("mov.u32 %0, %%smid;" : "=r" (smid));
  if (smid != SMID0 && smid != SMID1)
    return;
  
  if (smid == SMID0) {
    while (y == x) {
      for (ptr = (uint64_t *)page0[0]; ptr != page0; ptr = (uint64_t *)ptr[0])
        ++ptr[2];
      
      clk0 = clock64();
      clk1 = 0;
      while (clk1 < WAIT_TIME)
        clk1 = clock64() - clk0;
      
      y = ptr[1];
    }
  } else if (smid == SMID1) {
    while (y == x) {
      for (ptr = (uint64_t *)page1[0]; ptr != page1; ptr = (uint64_t *)ptr[0])
        ++ptr[2];
      
      y = ptr[1];
    }
  }
  
  page0[1] = 0;
  page1[1] = 0;
}

__global__ void
put(uint64_t *page, uint64_t x1, uint64_t x2)
{
  page[0] = x1;
  page[1] = x2;
}

int 
main(int argc, char *argv[])
{
  uint8_t *chunk0 = NULL;
  uint8_t *chunk1 = NULL;
  uint8_t *base = NULL;
  uint64_t *list0[PAGE0_NUM];
  uint64_t *list1[PAGE1_NUM];
  uint64_t *dummy = NULL;
  
  hipDeviceReset();
  hipFuncSetAttribute(reinterpret_cast<const void*>(loop), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM);
  
  // hoard a large address space
  hipMallocManaged(&chunk0, CHUNK0_SIZE);
  hipMallocManaged(&chunk1, CHUNK1_SIZE);
  
  base = (uint8_t *)BASE_ADDR;
  for (int i = 0; i < PAGE0_NUM; ++i)
    list0[i] = (uint64_t *)(base + i * STRIDE_SIZE);
  base += PAGE0_NUM * STRIDE_SIZE;
  for (int i = 0; i < PAGE1_NUM; ++i)
    list1[i] = (uint64_t *)(base + i * STRIDE_SIZE);
  dummy = (uint64_t *)DUMMY_ADDR;
  
  for (int i = 0; i < PAGE0_NUM; ++i)
    put<<<1, 1>>>(list0[i], (uint64_t)list0[(i + 1) % PAGE0_NUM], 0xdeadbeef);
  for (int i = 0; i < PAGE1_NUM; ++i)
    put<<<1, 1>>>(list1[i], (uint64_t)list1[(i + 1) % PAGE1_NUM], 0xdeadbeef);
  put<<<1, 1>>>(dummy, 0, 0);
  hipDeviceSynchronize();
  
  loop<<<BLK_NUM, 1, SHARED_MEM>>>(list0[0], list1[0], 0xdeadbeef);
  hipDeviceSynchronize();
  
  hipFree(chunk0);
  hipFree(chunk1);
}


