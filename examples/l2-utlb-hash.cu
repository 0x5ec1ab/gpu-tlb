#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

#define CHUNK0_SIZE (64L * 1024L * 1024L * 1024L * 1024L + 0x55554000000L)
#define CHUNK1_SIZE (41L * 1024L * 1024L * 1024L * 1024L + 0x0ffc8000000L)
#define STRIDE_SIZE (1L * 1024L * 1024L)

#define BASE_ADDR   0x700000000000
#define DUMMY_ADDR  0x7F0000000000

#define PAGE0_NUM   1500
#define PAGE1_NUM   4000
#define PAGE2_NUM   16
#define WAIT_TIME   10000000000L // about 5 seconds on RTX3080

#define BLK_NUM     100
#define SHARED_MEM  (96 * 1024)
#define SMID0       0
#define SMID1       3 // IMPORTANT: SM0 and SM3 are in different GPCs on RTX3080

// derived indices that goes into the same L2-uTLB set on RTX3080
int idx[] = {32, 161, 290, 419, 548, 677, 806, 935, 1064};

/*******************************************************************************
 * derive the L2-uTLB set selection function
 ******************************************************************************/
__global__ void 
loop(volatile uint64_t *page0, volatile uint64_t *page1, volatile uint64_t *page2, uint64_t x)
{
  uint64_t y = x;
  volatile uint64_t *ptr;
  volatile uint64_t *evt;
  uint64_t clk0 = 0;
  uint64_t clk1 = 0;
  uint32_t smid;
  
  asm("mov.u32 %0, %%smid;" : "=r" (smid));
  if (smid != SMID0 && smid != SMID1)
    return;
  
  if (smid == SMID0) {
    while (y == x) {
      for (ptr = (uint64_t *)page0[0]; ptr != page0; ptr = (uint64_t *)ptr[0]) {
        // access L1-dTLB eviction set
        for (evt = (uint64_t *)page2[0]; evt != page2; evt = (uint64_t *)evt[0])
          ++evt[2];
        ++ptr[2];
      }
      
      clk0 = clock64();
      clk1 = 0;
      while (clk1 < WAIT_TIME)
        clk1 = clock64() - clk0;
            
      y = ptr[1];
    }
  } else if (smid == SMID1) {
    while (y == x) {
      for (ptr = (uint64_t *)page1[0]; ptr != page1; ptr = (uint64_t *)ptr[0])
        ++ptr[2];
      
      y = ptr[1];
    }
  } 
  
  page0[1] = 0;
  page1[1] = 0;
}

__global__ void
put(uint64_t *page, uint64_t x1, uint64_t x2)
{
  page[0] = x1;
  page[1] = x2;
}

int 
main(int argc, char *argv[])
{
  uint8_t *chunk0 = NULL;
  uint8_t *chunk1 = NULL;
  uint8_t *base = NULL;
  uint64_t *list0[PAGE0_NUM];
  uint64_t *list1[PAGE1_NUM];
  uint64_t *list2[PAGE2_NUM];
  uint64_t *chain[sizeof(idx) / sizeof(int)];
  int num = 0;
  uint64_t *dummy = NULL;
  int bit0 = -1;
  int bit1 = -1;
  uint64_t mask0 = 0;
  uint64_t mask1 = 0;
  uint64_t addr = 0;
  
  hipDeviceReset();
  hipFuncSetAttribute(reinterpret_cast<const void*>(loop), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM);
  
  // hoard a large address space
  hipMallocManaged(&chunk0, CHUNK0_SIZE);
  hipMallocManaged(&chunk1, CHUNK1_SIZE);
  
  base = (uint8_t *)BASE_ADDR;
  for (int i = 0; i < PAGE0_NUM; ++i)
    list0[i] = (uint64_t *)(base + i * STRIDE_SIZE);
  base += PAGE0_NUM * STRIDE_SIZE;
  for (int i = 0; i < PAGE1_NUM; ++i)
    list1[i] = (uint64_t *)(base + i * STRIDE_SIZE);
  for (int i = 0; i < PAGE2_NUM; ++i)
    list2[i] = list0[idx[0] + i + 1];
  num = sizeof(idx) / sizeof(int);
  for (int i = 0; i < num; ++i)
    chain[i] = list0[idx[i]];
  dummy = (uint64_t *)DUMMY_ADDR;
  
  //++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
  // manipulate the last virtual address in chain
  bit0 = argc >= 2 ? atoi(argv[1]) : -1;
  mask0 = bit0 >= 0 ? 0x0000000000000001UL << bit0 : 0;
  bit1 = argc == 3 ? atoi(argv[2]) : -1;
  mask1 = bit1 >= 0 ? 0x0000000000000001UL << bit1 : 0;
  addr = (uint64_t)chain[num - 1];
  addr ^= mask0;
  addr ^= mask1;
  
  if (((uint8_t *)addr < chunk0 || (uint8_t *)addr >= chunk0 + CHUNK0_SIZE) && 
      ((uint8_t *)addr < chunk1 || (uint8_t *)addr >= chunk1 + CHUNK1_SIZE)) {
    printf("out of scope: try another setting (e.g., BASE_ADDR or target)\n");
    return -1;
  }
  
  // an address in the set is generated
  for (int i = 0; i < num - 1; ++i) {
    if (addr == (uint64_t)chain[i])
      return 0;
  }
  
  for (int i = 0; i < PAGE1_NUM; ++i) {
    if (addr == (uint64_t)list1[i]) {
      list1[i] = chain[num - 1];
      break;
    }
  }

  for (int i = 0; i < PAGE2_NUM; ++i) {
    if (addr == (uint64_t)list2[i]) {
      list2[i] = list0[idx[0] + PAGE2_NUM + 1];
      break;
    }
  }
  
  chain[num - 1] = (uint64_t *)addr;
  //++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
  for (int i = 0; i < num; ++i)
    put<<<1, 1>>>(chain[i], (uint64_t)chain[(i + 1) % num], 0xdeadbeef);
  for (int i = 0; i < PAGE1_NUM; ++i)
    put<<<1, 1>>>(list1[i], (uint64_t)list1[(i + 1) % PAGE1_NUM], 0xdeadbeef);
  for (int i = 0; i < PAGE2_NUM; ++i)
    put<<<1, 1>>>(list2[i], (uint64_t)list2[(i + 1) % PAGE2_NUM], 0xdeadbeef);
  put<<<1, 1>>>(dummy, 0, 0);
  hipDeviceSynchronize();
    
  loop<<<BLK_NUM, 1, SHARED_MEM>>>(chain[0], list1[0], list2[0], 0xdeadbeef);
  hipDeviceSynchronize();
  
  hipFree(chunk0);
  hipFree(chunk1);
}


